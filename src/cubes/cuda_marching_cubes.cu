#include "hip/hip_runtime.h"
#include "marching_cubes_common.hpp"
#include "cuda_marching_cubes.hpp"

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>

__device__ inline int calcCubeIndex(const FlatGridCell& cell, float isovalue) {
    int index = 0;
    for (int i = 0; i < 8; i++) {
        if (cell.data[i].v < isovalue) index |= (1 << i);
    }
    return index;
}

__device__ inline float3 interpolate(const FlatGridCellEle& e1, const FlatGridCellEle& e2, float isovalue) {
    float mu = (isovalue - e1.v) / (e2.v - e1.v);
    return make_float3(
        e1.p.x + mu * (e2.p.x - e1.p.x),
        e1.p.y + mu * (e2.p.y - e1.p.y),
        e1.p.z + mu * (e2.p.z - e1.p.z)
    );
}

__device__ inline void intersection_coords(
    const FlatGridCell& cell, float isovalue, int cubeIndex, float3 intersections[12]
) {
    int intersectionsKey = d_edgeTable[cubeIndex];

    for (int i = 0; i < 12; ++i) {
        if (intersectionsKey & (1 << i)) {
            const unsigned v1 = d_edgeToVertices[i][0];
            const unsigned v2 = d_edgeToVertices[i][1];
            intersections[i] = interpolate(cell.data[v1], cell.data[v2], isovalue);
        }
    }
}

__device__ inline void triangulate_cell_gpu(
    const FlatGridCell& cell, float isovalue,
    float3* outVerts, float3* outNormals, const int *offsets, int tid
) {
    int cubeIndex = calcCubeIndex(cell, isovalue);

    if (d_edgeTable[cubeIndex] == 0) return;
    float3 intersections[12];
    intersection_coords(cell, isovalue, cubeIndex, intersections);

    int base = offsets[tid] * 3;
    for (int i = 0; d_triTable[cubeIndex][i + 2] != -1; i += 3) {
        float3 p1 = intersections[d_triTable[cubeIndex][i]];
        float3 p2 = intersections[d_triTable[cubeIndex][i + 1]];
        float3 p3 = intersections[d_triTable[cubeIndex][i + 2]];

        float3 u = make_float3(p2.x - p1.x, p2.y - p1.y, p2.z - p1.z);
        float3 v = make_float3(p3.x - p1.x, p3.y - p1.y, p3.z - p1.z);

        float3 norm = normalize(cross(u, v));

        outVerts[base]     = p1;
        outVerts[base + 1] = p2;
        outVerts[base + 2] = p3;

        outNormals[base]     = norm;
        outNormals[base + 1] = norm;
        outNormals[base + 2] = norm;
        base += 3;
    }
}

__global__ void triangulate_flat_kernel(GPU_Grid grid, float isovalue, 
        float3* outVerts, float3* outNormals,
        const int *offsets){
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if(x >= grid.x - 1 || y >= grid.y - 1 || z >= grid.z - 1)
        return;

    float3 p = make_float3(x, y, z);

    FlatGridCell cell = {
        FlatGridCellEle{make_float3(p.x,     p.y,     p.z    ), grid(x,   y,   z  )},
        FlatGridCellEle{make_float3(p.x+1.f, p.y,     p.z    ), grid(x+1, y,   z  )},
        FlatGridCellEle{make_float3(p.x+1.f, p.y,     p.z+1.f), grid(x+1, y,   z+1)},
        FlatGridCellEle{make_float3(p.x,     p.y,     p.z+1.f), grid(x,   y,   z+1)},
        FlatGridCellEle{make_float3(p.x,     p.y+1.f, p.z    ), grid(x,   y+1, z  )},
        FlatGridCellEle{make_float3(p.x+1.f, p.y+1.f, p.z    ), grid(x+1, y+1, z  )},
        FlatGridCellEle{make_float3(p.x+1.f, p.y+1.f, p.z+1.f), grid(x+1, y+1, z+1)},
        FlatGridCellEle{make_float3(p.x,     p.y+1.f, p.z+1.f), grid(x,   y+1, z+1)}
    };

    triangulate_cell_gpu(cell, isovalue, outVerts, outNormals, offsets, grid.index_g(p.x,p.y,p.z));
}

__global__ void count_triangles_kernel(GPU_Grid grid, float isovalue, int *outCounts){

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    
    if(x >= grid.x - 1 || y >= grid.y - 1 || z >= grid.z - 1)
        return;
    
    int cubeIndex = 0;
    if (grid(x,   y,   z  ) < isovalue) cubeIndex |= 1;
    if (grid(x+1, y,   z  ) < isovalue) cubeIndex |= 2;
    if (grid(x+1, y,   z+1) < isovalue) cubeIndex |= 4;
    if (grid(x,   y,   z+1) < isovalue) cubeIndex |= 8;
    if (grid(x,   y+1, z  ) < isovalue) cubeIndex |= 16;
    if (grid(x+1, y+1, z  ) < isovalue) cubeIndex |= 32;
    if (grid(x+1, y+1, z+1) < isovalue) cubeIndex |= 64;
    if (grid(x,   y+1, z+1) < isovalue) cubeIndex |= 128;

    int count = 0;
    for (int i = 0; d_triTable[cubeIndex][i] != -1; i += 3)
        ++count;

    outCounts[grid.index_g(x,y,z)] = count;
}

namespace CudaMarchingCubes
{
    void trinagulate_grid_flat(const Grid<float> &grid, float isovalue, 
        std::vector<glm::vec3>& outVerts, std::vector<glm::vec3>& outNormals){

        hipMemcpyToSymbol(HIP_SYMBOL(d_edgeToVertices), edgeToVerticesU, sizeof(edgeToVerticesU));
        hipMemcpyToSymbol(HIP_SYMBOL(d_edgeTable), edgeTable, sizeof(edgeTable));
        hipMemcpyToSymbol(HIP_SYMBOL(d_triTable), triTable, sizeof(triTable));
            
        const glm::vec3 size = grid.getSize();
        size_t numEle = size.x * size.y * size.z;
        const int totalCells = (size.x - 1) * (size.y - 1) * (size.z - 1);

        dim3 threads(16,16,4);
        dim3 blocks(
            ceil((double)(size.x - 1) / threads.x),
            ceil((double)(size.y - 1) / threads.y),
            ceil((double)(size.z - 1) / threads.z)
        );

        std::cout << blocks.x << " " << blocks.y << " " << blocks.z << std::endl;
        std::cout << blocks.x * threads.x << " " << blocks.y * threads.y << " " << blocks.z * threads.z << std::endl;
        std::cout << size.x - 1 << " " << size.y - 1 << " " << size.z - 1 << std::endl;

        float *d_data;
        hipMalloc(&d_data, numEle * sizeof(float));
        hipMemcpy(d_data, grid.vector_data(), numEle * sizeof(float), hipMemcpyHostToDevice);

        GPU_Grid d_grid(d_data, size.x, size.y, size.z);

        thrust::device_vector<int> d_counts(totalCells + 1);
        thrust::device_vector<int> d_offsets(totalCells + 1);

        count_triangles_kernel<<<blocks, threads>>>(d_grid, isovalue, thrust::raw_pointer_cast(d_counts.data()));
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "[count_triangles] CUDA kernel error: " << hipGetErrorString(err) << std::endl;
            return;
        }
        
        // prefix sum
        thrust::exclusive_scan(d_counts.begin(), d_counts.end(), d_offsets.begin());
        
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "[exclusive_scan] CUDA kernel error: " << hipGetErrorString(err) << std::endl;
            return;
        }

        int triNum;
        hipMemcpy(&triNum, thrust::raw_pointer_cast(d_offsets.data()+totalCells), sizeof(int), hipMemcpyDeviceToHost);
        const int vertNum = 3 * triNum;

        float3 *d_verts;
        float3* d_normals;
        hipMalloc(&d_verts, vertNum * sizeof(float3));
        hipMalloc(&d_normals, vertNum * sizeof(float3));


        triangulate_flat_kernel<<<blocks, threads>>>(d_grid, isovalue, d_verts, d_normals, thrust::raw_pointer_cast(d_offsets.data()));
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "[triangulate] CUDA kernel error: " << hipGetErrorString(err) << std::endl;
            return;
        }

        outVerts.resize(vertNum);
        outNormals.resize(vertNum);

        hipMemcpy(outVerts.data(), d_verts, vertNum * sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(outNormals.data(), d_normals, vertNum * sizeof(float3), hipMemcpyDeviceToHost);

        hipFree(d_data);
        hipFree(d_verts);
        hipFree(d_normals);
        // hipDeviceReset();
    }

    void trinagulate_grid(const Grid<float> &grid, float isovalue, 
        std::vector<glm::vec3>& outVerts, std::vector<glm::vec3>& outNormals){

        }
}